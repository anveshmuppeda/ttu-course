
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

//Computing the matrix multiplication
__global__ void gpu_Matrix_Multiplication(double *x, double *y, double *ans, int N) 
{
  //Storing index value with the thread ID, since we are using single block.
  int index = threadIdx.x;
  //Calculating stride value with the help of block dimension, to loop throgh the all elements in array
  int stride = blockDim.x;
  //Total number of elemenst(operations) for result matrix
  int total = N*N;
  
  for (int i = index; i < total; i += stride) 
  {
      for(int k = 0; k < N; k++) {
        //Calculating the result matrix value
        ans[i] += ( x[k*index+k] * y[ index * N + k ] );
      }
  }
}

// ---------------------------------------------------------------------- check function
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    // size of matrix
    int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
    //Loopig the kernel trough iter
    int iter = 3;
    clock_t t;

    // Martices
    double *x, *y, *ans;
    
    // Allocate Unified Memory - accessible from both CPU and GPU
    hipMallocManaged(&x, N*N*sizeof(double));
    hipMallocManaged(&y, N*N*sizeof(double));
    hipMallocManaged(&ans, N*N*sizeof(double));

    // initialize x,y and ans arrays on the host
    for (int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
        }
    }

    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        //Launch kernel with blocks and blocksize
        gpu_Matrix_Multiplication<<<1, 512>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;  //we will ignore the first run
        // printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
    }

    //Calculating the avg time over the 3 iterations
    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    //Priting the Average value i.e. avg time to compute GPU kernel
    printf("It took %lf ms on avg.\n", avg);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);

    return 0;
}
/* EOF */