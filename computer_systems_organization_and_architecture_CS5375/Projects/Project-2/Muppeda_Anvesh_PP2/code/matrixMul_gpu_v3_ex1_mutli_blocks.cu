
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

//Computing the matrix multiplication
__global__ void gpu_Matrix_Multiplication(double *x, double *y, double *ans, int N) 
{
  //Calculate the index value based on th threadId and blockId values(here we are passing multiple blocks along with the multiple threads)
  int index = blockIdx.x * N + threadIdx.x;
  //Iterate over row, and column of the target matrices
  for (int k = 0; k < N; k++) {
    //Accumulate results for a single element
    ans[ index ] += x[ blockIdx.x * N + k] * y[k * N + threadIdx.y ];
  }
}

// ---------------------------------------------------------------------- check function on host
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    // size of matrix
    int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
    int iter = 3;
    clock_t t;

    // Martices
    double *x, *y, *ans;
    
    // Allocate Unified Memory - accessible from both CPU and GPU
    hipMallocManaged(&x, N*N*sizeof(double));
    hipMallocManaged(&y, N*N*sizeof(double));
    hipMallocManaged(&ans, N*N*sizeof(double));

    // initialize x,y and ans arrays on the host
    for (int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
        }
    }

    // Run kernel on N*N elements on the GPU
    //Defining the Block Size
    int blockSize = 512;
    //Defining the number of blocks
    int numBlocks = (N * N) / blockSize;

    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        // Launch kernel with blocks and blocksize
        gpu_Matrix_Multiplication<<<numBlocks, blockSize>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;  //we will ignore the first run
        // printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
    }

    //Calculating the avg time over the 3 iterations
    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    //Priting the Average value i.e. avg time to compute GPU kernel
    printf("It took %lf ms on avg.\n", avg);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);

    return 0;
}
/* EOF */