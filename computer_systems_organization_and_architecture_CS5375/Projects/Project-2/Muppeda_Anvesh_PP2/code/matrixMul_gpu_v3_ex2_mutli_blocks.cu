
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

__global__ void gpu_Matrix_Multiplication(double *x, double *y, double *ans, int N) 
{
  //calculates current thread ID with the help of block and grid dim
	int t = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x);
	//calculates current block id with the help of grid and block id
	int b = (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
	//block size (Number of threads per block)
	int T = blockDim.x*blockDim.y*blockDim.z;
	//grid size (Number of blocks per grid)
	int B = gridDim.x*gridDim.y*gridDim.z;

  //Calculating the AC value with the above values
  int AC = (N*N)/(T*B);
  int threadComputation = AC * N;
  /*  printf("Curretly total number of Assigned Cells are: %d\n", AC);
      printf("Each cell in the matrix is assigned to a different thread. 
      Each thread do O(%d) computation.", threadComputation);
	    Assigned cells of different threads does not overlape with 
      each other. And so no need for synchronization.
	*/

    for (int i=b;i<N;i+=B)
    {
      for(int j=t;j<N;j+=T)
      {
        for(int k=0;k<N;k++)        
        {
          //Calculating the target matrix
          ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
        }
      }
	  }
}

// ---------------------------------------------------------------------- check function on the host
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    // size of matrix
    int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
    int iter = 3;
    clock_t t;

    // Martices
    double *x, *y, *ans;
    
    // Allocate Unified Memory - accessible from both CPU and GPU
    hipMallocManaged(&x, N*N*sizeof(double));
    hipMallocManaged(&y, N*N*sizeof(double));
    hipMallocManaged(&ans, N*N*sizeof(double));

    //Initialize x,y and ans arrays on the host
    for (int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
        }
    }
    
    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        //Launch kernel on GPU with the 1024 blocks and 1024 threads
        gpu_Matrix_Multiplication<<<dim3(64,4,4), dim3(64,4,4)>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;  //we will ignore the first run
        // printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
    }

    //Calculating the avg time over the 3 iterations
    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    //Priting the Average value i.e. avg time to compute GPU kernel
    printf("It took %lf ms on avg.\n", avg);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);

    return 0;
}
/* EOF */