
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

__global__ void gpu_Matrix_Multiplication(double *x, double *y, double *ans, int N) 
{
  //calculates current thread ID with the help of block and grid dim
	int t = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x);
	//calculates current block id with the help of grid and block id
	int b = (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
	//block size (Number of threads per block)
	int T = blockDim.x*blockDim.y*blockDim.z;
	//grid size (Number of blocks per grid)
	int B = gridDim.x*gridDim.y*gridDim.z;
	
  //Calculating the AC value with the above values
  int AC = (N*N)/(T*B);
  int threadComputation = AC * N;
  /*  printf("Curretly total number of Assigned Cells are: %d\n", AC);
      printf("Each cell in the matrix is assigned to a different thread. 
      Each thread do O(%d) computation.", threadComputation);
	    Assigned cells of different threads does not overlape with 
      each other. And so no need for synchronization.
	 */
	 
    for (int i=b;i<N;i+=B)
    {
      for(int j=t;j<N;j+=T)
      {
        for(int k=0;k<N;k++)
        {
          //Calculating the target matrix
          ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
        }
      }
	  }
}

//initialize x,y and ans arrays on the GPU Device
__global__ void init(double *x, double *y, double *ans, int N) 
{
  //Caluculating the row with the help of block and thread IDs
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  //Calculating the column with the help of block an thread IDs
  int j = blockIdx.x * blockDim.x + threadIdx.x;
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
}

// ---------------------------------------------------------------------- check function on Host
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    int N= 1<<9; //Matrix Size
    int iter = 3;
    clock_t t;

    // allocate memory in host RAM
    double *x, *y, *ans;
    hipMallocManaged((void **) &x, sizeof(double)*N*N);
    hipMallocManaged((void **) &y, sizeof(double)*N*N);
    hipMallocManaged((void **) &ans, sizeof(double)*N*N);

    //Defining the number of threads per block
    int THREADS = 8;

    //Defining the number of blocks to the grid
    int BLOCKS = N / THREADS;

    //Calculating the number of threads and blocks using the 2D dim3
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);
   
    // Launch initilization kernel with the 4095 blocks and 64 threads
    init<<<blocks, threads>>>(x, y, ans, N);
    
    // Launch matrix multiplcation kernel
    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        //Launch the kernel with the 256 threads and 256 blocks which is optimized
        gpu_Matrix_Multiplication<<<dim3(2,2,64), dim3(2,2,64)>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;  //we will ignore the first run
        // printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
    }

    //Calculating the avg time over the 3 iterations
    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    //Priting the Average value i.e. avg time to compute GPU kernel
    printf("It took %lf ms on avg.\n", avg);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);

    return 0;
}
/* EOF */