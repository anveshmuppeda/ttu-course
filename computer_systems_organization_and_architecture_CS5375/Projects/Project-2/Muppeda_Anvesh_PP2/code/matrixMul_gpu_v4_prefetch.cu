#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

__global__ void gpu_Matrix_Multiplication(double *x, double *y, double *ans, int N) 
{
  //calculates unique thread ID in the block
	int t= (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x);
	//calculates unique block ID in the grid
	int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
	//block size (this is redundant though)
	int T= blockDim.x*blockDim.y*blockDim.z;
	//grid size (this is redundant though)
	int B= gridDim.x*gridDim.y*gridDim.z;
	
  int AC = (N*N)/(T*B);
  int threadComputation = AC * N;
  /*  printf("Curretly total number of Assigned Cells are: %d\n", AC);
      printf("Each cell in the matrix is assigned to a different thread. 
      Each thread do O(%d) computation.", threadComputation);
	    Assigned cells of different threads does not overlape with 
      each other. And so no need for synchronization.
	 */
	 
    for (int i=b;i<N;i+=B)
    {
      for(int j=t;j<N;j+=T)
      {
        for(int k=0;k<N;k++)
        {
          ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
        }
      }
	  }
}

//initialize x,y and ans arrays on the GPU
__global__ void init(double *x, double *y, double *ans, int N) 
{
  //Caluculating the row
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  //Calculating the column
  int j = blockIdx.x * blockDim.x + threadIdx.x;
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
}

// ---------------------------------------------------------------------- check
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    int N= 1<<9;
    int iter = 3;
    clock_t t;

    // allocate memory in host RAM
    double *x, *y, *ans;
    hipMallocManaged((void **) &x, sizeof(double)*N*N);
    hipMallocManaged((void **) &y, sizeof(double)*N*N);
    hipMallocManaged((void **) &ans, sizeof(double)*N*N);

    // Prefetch the data to the GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*N*sizeof(double), device, NULL);
    hipMemPrefetchAsync(y, N*N*sizeof(double), device, NULL);
    hipMemPrefetchAsync(ans, N*N*sizeof(double), device, NULL);

    //Defining the number of threads per block
    int THREADS = 8;

    //Defining the number of blocks to the grid
    int BLOCKS = N / THREADS;

    //Calculating the number of threads and blocks using the 2D dim3
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);
   
    // Launch initilization kernel with the 4095 blocks and 64 threads
    init<<<blocks, threads>>>(x, y, ans, N);
    
    // Launch matrix multiplcation kernel
    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        //Launch the kernel with the 256 threads and 256 blocks which is optimized
        gpu_Matrix_Multiplication<<<dim3(2,2,64), dim3(2,2,64)>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;  //we will ignore the first run
        // printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
    }

    //Calculating the avg time over the 3 iterations
    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    //Priting the Average value i.e. avg time to compute GPU kernel
    printf("It took %lf ms on avg.\n", avg);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);

    return 0;
}
/* EOF */